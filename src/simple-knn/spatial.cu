#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  // _xyz 的大小为 [9443, 3]，所以 points.size(0) 返回的是 9443，也就是说：
  //  P = 9443;
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  // 这行代码创建一个一维张量 means，长度为 P，并初始化为全0，数据类型为 
  torch::Tensor means = torch::full({P}, 0.0, float_opts);
  
  // 代码调用 SimpleKNN::knn 函数，计算每个点到最近点的平方距离，并将结果存储在 means 张量中。
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}